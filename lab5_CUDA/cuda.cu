#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

#include <cv.h>
#include <highgui.h>
#include <time.h>
#include <sys/time.h>

using namespace cv;

#define ARRAY_SIZE 6220800
#define COLS 5760
#define ARRAY_SIZE_IN_BYTE (sizeof(unsigned char)*(ARRAY_SIZE))

__global__ void compute(uchar *dev_src,uchar *dev_dst)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	int thread_idx = idx + idy* blockDim.x * gridDim.x;
	//仍然忽略边界	
	//这里没有考虑计算的边界，但是也没有出现像段错误的那种错误？？？？？
	//if(idy!=0&&idy!=1079&&idx>2&&idx<5760-3)
	*(dev_dst+thread_idx) = dev_src[thread_idx-COLS-3]+dev_src[thread_idx-COLS]+dev_src[thread_idx-COLS+3]+\
							dev_src[thread_idx]*(-7)+dev_src[thread_idx-3]+dev_src[thread_idx+3]+\
							dev_src[thread_idx+COLS-3]+dev_src[thread_idx+COLS]+dev_src[thread_idx+COLS+3];
}


int main(void)
{
	
	const char* imageName = "../TestPicture.jpg";
	Mat image;
	image = imread( imageName, CV_LOAD_IMAGE_COLOR);

	if( image.empty()==true)
	{
		printf( " No image data \n " );
		return -1;
	}
	const int nChannels = image.channels();
	Mat con_image;
	con_image.create(image.size(),image.type());
	struct timeval start,finish;
	double costTime;
	//start time
	gettimeofday(&start,0);
	
	dim3 grid(81, 150);
	dim3 block(128, 4);	

	uchar *dev_dst;
	uchar *dev_src;
	hipMalloc((void**)&dev_dst, ARRAY_SIZE_IN_BYTE);
	hipMalloc((void**)&dev_src, ARRAY_SIZE_IN_BYTE);
		
	hipMemcpy(dev_src, image.ptr<uchar>(0), ARRAY_SIZE_IN_BYTE, hipMemcpyHostToDevice);
	compute<<<grid, block>>>(dev_src,dev_dst);
	hipMemcpy(con_image.ptr<uchar>(0), dev_dst, ARRAY_SIZE_IN_BYTE, hipMemcpyDeviceToHost);

	
	con_image.row(0).setTo(Scalar(0));
	con_image.row(con_image.rows-1).setTo(Scalar(0));
    con_image.col(0).setTo(Scalar(0));
    con_image.col(con_image.cols - 1).setTo(Scalar(0));
	//finish time
	gettimeofday(&finish,0);
	costTime = 1000000*(finish.tv_sec-start.tv_sec)+finish.tv_usec-start.tv_usec;
	printf("cost time:%lf us.\n",costTime);
	IplImage qImg;
	qImg = IplImage(con_image);
	cvSaveImage("./con.jpg", &qImg);
	
	hipFree(dev_src);
	hipFree(dev_dst);
	return 0;
}

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
//#include <conio.h>

#define ARRAY_SIZE 80
#define ARRAY_SIZE_IN_BYTE (sizeof(unsigned int)*(ARRAY_SIZE))

__global__ void what_is_my_id(unsigned int *const block,
	unsigned int *const thread,
	unsigned int *const warp,
	unsigned int *const calc_thread)
{
	//int x = threadIdx.x + threadIdx.y*blockDim.x;
	//int y = blockIdx.x + blockIdx.y*gridDim.x;
	//const unsigned int thread_idx = x + y*gridDim.x*blockDim.x;
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	int thread_idx = idx + idy* blockDim.x * gridDim.x;

	block[thread_idx] = blockDim.y;
	thread[thread_idx] = idy;
	calc_thread[thread_idx] = thread_idx;
}

unsigned int cpu_block[ARRAY_SIZE];
unsigned int cpu_thread[ARRAY_SIZE];
unsigned int cpu_warp[ARRAY_SIZE];
unsigned int cpu_calc_thread[ARRAY_SIZE];

int main(void)
{
	dim3 grid(2, 5);
	dim3 block(2, 4);
	char ch;

	unsigned int *gpu_block;
	unsigned int *gpu_thread;
	unsigned int *gpu_warp;
	unsigned int *gpu_calc_thread;

	unsigned int i;

	hipMalloc((void**)&gpu_block, ARRAY_SIZE_IN_BYTE); //需要两层指针，修改gpu_block的值
	hipMalloc((void**)&gpu_thread, ARRAY_SIZE_IN_BYTE);
	hipMalloc((void**)&gpu_warp, ARRAY_SIZE_IN_BYTE);
	hipMalloc((void**)&gpu_calc_thread, ARRAY_SIZE_IN_BYTE);

	what_is_my_id << <grid, block >> >(gpu_block, gpu_thread, gpu_warp, gpu_calc_thread);

	hipMemcpy(cpu_block, gpu_block, ARRAY_SIZE_IN_BYTE, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_thread, gpu_thread, ARRAY_SIZE_IN_BYTE, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_warp, gpu_warp, ARRAY_SIZE_IN_BYTE, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_calc_thread, gpu_calc_thread, ARRAY_SIZE_IN_BYTE, hipMemcpyDeviceToHost);

	hipFree(gpu_block);
	hipFree(gpu_thread);
	hipFree(gpu_warp);
	hipFree(gpu_calc_thread);

	for (int i = 0; i<ARRAY_SIZE; i++)
	{
		printf("Calculated Thread:%3u - Block:%2u - Warp %2u - Thread %3u\n", cpu_calc_thread[i], cpu_block[i], cpu_warp[i], cpu_thread[i]);
	}
	ch = getchar();
	return 0;
}
